#include "hip/hip_runtime.h"
#ifdef OS_WINDOWS
#include "hip/hip_runtime.h"
#include ""
#define THREADS_PER_BLOCK 32 //1024 total threads per block
#else
#define THREADS_PER_BLOCK 16 //512 total threads per block
#include <sys/time.h>
#endif

#include <stdlib.h> 
#include <stdio.h> 

#define DELTA_X 1024 //This is a square of 1024
#define DELTA_Y 1024 //    by 1024

#define MAX_STEPS_KERNEL 10 //number of computations in kernel per cycle
#define MIN_VARIATION 0.05

__device__ int step = 0;

__global__ void calc_diffuse(double *current_mat, double *prev_mat, size_t rows, size_t cols, int *partial_result)
{
	if (step == MAX_STEPS_KERNEL){
		if (blockIdx.x == 0 && threadIdx.x == 0) {
			*partial_result = 1;
		}
		return;
	} else {
		int col = blockIdx.x*blockDim.x + threadIdx.x;
		int row = blockIdx.y*blockDim.y + threadIdx.y;

		if ((row < rows) && (col < cols)) {
			if (row == (rows/2) && col == (cols/2)){
				current_mat[(row)*cols + (col)] = 100;
			}
			else if (row == 0 && col == 0) {
				current_mat[(row)*cols + (col)] = (23 + prev_mat[(row + 1)*cols + (row)] + 23 + prev_mat[(row)*cols + (col + 1)]) / 4;
			}
			else if (row == 0 && col <= cols - 1) {
				current_mat[(row)*cols + (col)] = (23 + prev_mat[(row + 1)*cols + (row)] + prev_mat[(row)*cols + (col - 1)] + prev_mat[(row)*cols + (col + 1)]) / 4;
			}
			else if (row == 0 && col == cols - 1) {
				current_mat[(row)*cols + (col)] = (23 + prev_mat[(row + 1)*cols + (row)] + prev_mat[(row)*cols + (col - 1)] + 23) / 4;
			}
			else if (row < rows -1 && col == 0) {
				current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + prev_mat[(row + 1)*cols + (row)] + 23 + prev_mat[(row)*cols + (col + 1)]) / 4;
			}
			else if (row == rows-1 && col == 0) {
				current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + 23 + 23 + prev_mat[(row)*cols + (col + 1)]) / 4;
			}
			else if (row == rows - 1 && col < cols - 1) {
				current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + 23 + prev_mat[(row)*cols + (col - 1)] + prev_mat[(row)*cols + (col + 1)]) / 4;
			}
			else if (row == rows - 1 && col == cols - 1) {
				current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + 23 + prev_mat[(row)*cols + (col - 1)] + 23) / 4;
			}
			else {
				current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + prev_mat[(row + 1)*cols + (row)] + prev_mat[(row)*cols + (col - 1)] + prev_mat[(row)*cols + (col + 1)]) / 4;
			}
			atomicAdd(&step, 1);
			
		}
	}
}

/*__global__ void debug_step(double *mat, size_t cols) {
	int new_step = step;
	int row = threadIdx.y;
	int col = threadIdx.x;
	double tmp = mat[(row)*cols + (col)];
}*/

void cudasafe(hipError_t error, char* message)
{
	if (error != hipSuccess) {
		fprintf(stderr, "ERROR: %s : %i\n", message, error); exit(-1);
	}

}

int init_t0(double *mat1, size_t rows, size_t cols, double K, double room_temp)
{
	int i;
	int j;
	for (i = 0; i < rows; i++) {
		for (j = 0; j < cols; j++) {
			mat1[i*cols + j] = room_temp;
		}
	}
	mat1[(rows/2)*cols + (cols/2)] = K;
	return 0;
}

bool sequential_diffuse(double *current_mat, double *prev_mat, size_t rows, size_t cols, size_t *steps){
	unsigned int is_finished = 1;
	int col = 0;
        int row = 0;
        for (row = 0; row < rows; row++){
		for(col=0; col < cols; col++){
			if (row == (rows/2) && col == (cols/2)){
                                current_mat[(row)*cols + (col)] = 100;
                        }
			else if (row == 0 && col == 0) {
        	        	current_mat[(row)*cols + (col)] = (100 + prev_mat[(row + 1)*cols + (row)] + 100 + prev_mat[(row)*cols + (col + 1)]) / 4;
                	}
                	else if (row == 0 && col <= cols - 1) {
                		current_mat[(row)*cols + (col)] = (23 + prev_mat[(row + 1)*cols + (row)] + prev_mat[(row)*cols + (col - 1)] + prev_mat[(row)*cols + (col + 1)]) / 4;
                	}
                	else if (row == 0 && col == cols - 1) {
	        	       	current_mat[(row)*cols + (col)] = (23 + prev_mat[(row + 1)*cols + (row)] + prev_mat[(row)*cols + (col - 1)] + 23) / 4;
                	}
                	else if (row < rows -1 && col == 0) {
        		        current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + prev_mat[(row + 1)*cols + (row)] + 23 + prev_mat[(row)*cols + (col + 1)]) / 4;
                	}
                	else if (row == rows-1 && col == 0) {
                		current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + 23 + 23 + prev_mat[(row)*cols + (col + 1)]) / 4;
                	}
                	else if (row == rows - 1 && col < cols - 1) {
                		current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + 23 + prev_mat[(row)*cols + (col - 1)] + prev_mat[(row)*cols + (col + 1)]) / 4;
                	}
                	else if (row == rows - 1 && col == cols - 1) {
                		current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + 23 + prev_mat[(row)*cols + (col - 1)] + 23) / 4;
                	}
               	 	else {
                		current_mat[(row)*cols + (col)] = (prev_mat[(row - 1)*cols + (col)] + prev_mat[(row + 1)*cols + (row)] + prev_mat[(row)*cols + (col - 1)] + prev_mat[(row)*cols + (col + 1)]) / 4;
                	}
		}
	}
	
	for (row = 0; row < rows; row++){
                for(col=0; col < cols; col++){
			unsigned int stop = 0;
			if(current_mat[(row)*cols + (col)] > prev_mat[(row)*cols + (col)]-MIN_VARIATION && current_mat[(row)*cols + (col)] < prev_mat[(row)*cols + (col)]+MIN_VARIATION) {
                                stop  = 1;
                        }
			is_finished = is_finished&stop;
		}
	}
	*steps=*steps + 1;
	//printf("From the seq: stops?: %d ",is_finished);
     
	if(is_finished == 1){
		return true;
	}
        return false;

}

void write_matrix(double *mat, size_t rows, size_t cols, char* file_name) {
	FILE *f = fopen(file_name, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}
	int i;
	int j;
	fprintf(f, "{\"z\": [");
	for (i = 0; i < rows; i++) {
		fprintf(f, "[");
		for (j = 0; j < cols; j++) {
			if (j == cols - 1) {
				fprintf(f, " %.5f ", mat[i*cols + j]);
			}
			else {
				fprintf(f, " %.5f, ", mat[i*cols + j]);
			}
			
		}
		if (i == rows - 1) {
			fprintf(f, "]\n");
		}
		else {
			fprintf(f, "],\n");
		}
		
	}
	fprintf(f, "] }");
	fclose(f);
}

struct timeval  tp1, tp2;

int main(int argc, char *argv[]) 
{ 
	long total_steps = 0;
  	// setup/initialize
  	if (argc != 2) {
    		printf ("usage: progName <steps>\n");
    		exit(-1);
  	} else {
    		total_steps = atol(argv[1]);
	}

	double *mat1;
	double *mat2;
	
	mat1 = (double*)calloc(DELTA_X * DELTA_Y,sizeof(double));
	mat2 = (double*)calloc(DELTA_X * DELTA_Y,sizeof(double));
	
	init_t0(mat1, DELTA_X, DELTA_Y, 1000.0, 23.0);

	//sequential_diffuse
        gettimeofday(&tp1, NULL);
	printf("Starting sequential\n");
	bool stop = false;
        bool is_swap = false;
	size_t seq_steps = 0;
	while(seq_steps < total_steps){
	//while(!stop){
		if(is_swap){
			stop = sequential_diffuse(mat1, mat2, DELTA_X, DELTA_Y, &seq_steps);
		} 
		else {
			stop = sequential_diffuse(mat2, mat1, DELTA_X, DELTA_Y, &seq_steps);
		}
		//printf("Step %d \n",seq_steps);
		is_swap = !is_swap;
	}
	gettimeofday(&tp2, NULL);
	double seq_time_result = (double) (tp2.tv_usec - tp1.tv_usec) / 1000000 + (double) (tp2.tv_sec - tp1.tv_sec);

	printf("Sequential finished in %d steps, writiing to file\n",seq_steps);
	// Allocates storage
	char *file_name = (char*)malloc(13 * sizeof(char));
	// Prints "Hello world!" on hello_world
	sprintf(file_name, "./seq_data_t%d.json",total_steps);
	if(is_swap){
		write_matrix(mat1, DELTA_X, DELTA_Y, file_name);
	}else{
		write_matrix(mat2, DELTA_X, DELTA_Y, file_name);
	}

	double *mat1_d, *mat2_d;
	int *result_d;
	printf("Allocating CUDA memory \n");

	init_t0(mat1, DELTA_X, DELTA_Y, 1000.0, 23.0);
	gettimeofday(&tp1, NULL);

	cudasafe(hipMalloc ((void**) &result_d, sizeof(int)), "hipMalloc(result) failed.");
	cudasafe(hipMemcpy (result_d,&stop, sizeof(int), hipMemcpyHostToDevice), "hipMemcpy host->dev (result) failed.");

	cudasafe(hipMalloc((void**)&mat1_d, DELTA_X * DELTA_Y * sizeof(double)), "hipMalloc(mat1_d) failed.");
	cudasafe(hipMemcpy(mat1_d, mat1, DELTA_X * DELTA_Y * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy host->dev (mat1_d) failed.");

	cudasafe(hipMalloc((void**)&mat2_d, DELTA_X * DELTA_Y * sizeof(double)), "hipMalloc(mat2_d) failed.");
	cudasafe(hipMemcpy(mat2_d, mat2, DELTA_X * DELTA_Y * sizeof(double), hipMemcpyHostToDevice), "hipMemcpy host->dev (mat2_d) failed.");

	long long max_heap_size_d = (long long)3*1024*1024*1024;
	hipDeviceSetLimit(hipLimitMallocHeapSize, max_heap_size_d); 
 
	// set execution configuration
	long long block_size = THREADS_PER_BLOCK;
	long long blocks_x = ceil(DELTA_X / ((float) block_size));
	long long blocks_y = ceil(DELTA_Y / ((float) block_size));
	dim3 dimblock (THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	dim3 dimgrid (blocks_x, blocks_y);
	printf("Starting Kernel \n");
	unsigned int time_steps;
	for (time_steps = 0; time_steps < seq_steps; time_steps++) {
		if(time_steps%2 == 0){
			calc_diffuse << < dimgrid, dimblock >> > (mat2_d, mat1_d, DELTA_X, DELTA_Y, result_d);
		}
		else {
			calc_diffuse << < dimgrid, dimblock >> > (mat1_d, mat2_d, DELTA_X, DELTA_Y, result_d);
		}
		
		//debug_step <<<1, 1 >>> (mat2_d, DELTA_X);
	}

	printf("Kernel Finished \n");
	cudasafe(hipMemcpy(mat1, mat1_d, DELTA_X * DELTA_Y * sizeof(double), hipMemcpyDeviceToHost), "hipMemcpy host <-dev(mat1_d) failed.");
	cudasafe(hipMemcpy(mat2, mat2_d, DELTA_X * DELTA_Y * sizeof(double), hipMemcpyDeviceToHost), "hipMemcpy host <-dev(mat2_d) failed.");

	gettimeofday(&tp2, NULL);

	double par_time_result = (double) (tp2.tv_usec - tp1.tv_usec) / 1000000 + (double) (tp2.tv_sec - tp1.tv_sec);
        
	hipDeviceSynchronize();
	printf("Writing to file \n");
	sprintf(file_name, "./par_data_t%d.json",total_steps);
	if(is_swap){
                write_matrix(mat1, DELTA_X, DELTA_Y, file_name);
        }else{
                write_matrix(mat2, DELTA_X, DELTA_Y, file_name);
        }
	printf("Finished writing \n");
	
	printf("Problem size: %d \n",DELTA_X*DELTA_Y);
	printf("seq_time, %.5f\n", seq_time_result);
	printf("par_time, %.5f\n", par_time_result);
	printf("speed_up, %.5f\n", seq_time_result/par_time_result);
	//printf("Press any key to exit!");
	//getchar();
 
	return 0; 
}
